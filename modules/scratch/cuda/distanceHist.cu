#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Example of integrating CUDA functions into an existing
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <IMP/scratch/internal/hip/hip_runtime_api.h>
#include <IMP/scratch/internal/helper_functions.h>

#define SQ(x) (x)*(x)


__constant__ int DN; // number of atoms
__constant__ float DDRH; // histogram resolution

__global__ void gpu_histogram_kernel(float3 *in, float *nhis) {

  int iBlockBegin = (DN/gridDim.x)*blockIdx.x;
  int iBlockEnd = min((DN/gridDim.x)*(blockIdx.x+1),DN);
  int jBlockBegin = (DN/gridDim.y)*blockIdx.y;
  int jBlockEnd = min((DN/gridDim.y)*(blockIdx.y+1),DN);
  for (int i=iBlockBegin+threadIdx.x; i<iBlockEnd; i+=blockDim.x) {
    float3 a = in[i];
    for (int j=jBlockBegin+threadIdx.y; j<jBlockEnd; j+=blockDim.y) {
      if (i<j) {
        // Process (i,j) atom pair
        float3 b = in[j];
        float rij = sqrt(SQ(a.x - b.x) + SQ(a.y - b.y) + SQ(a.z - b.z));
        int ih = rij/DDRH;
        // nhis[ih] += 1.0; /* Entry to the histogram */
        atomicAdd(&nhis[ih],1.0);
      } // end if i<j
    } // end for j
  } // end for i
}


extern "C" void
get_distance_histogram_gpu(int argc, char ** argv, int len, float3 *coords, int hist_size, float *nhis)
{
  // use command-line specified CUDA device, otherwise use device with highest Gflops/s
  findCudaDevice(1, (const char **)argv);

  float delta = 0.5;
  hipMemcpyToSymbol(HIP_SYMBOL(DN), &len, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(DDRH), &delta, sizeof(float), 0, hipMemcpyHostToDevice);


  // allocate device memory
  float3 *dev_data; // atomic positions
  checkCudaErrors(hipMalloc((void **) &dev_data, sizeof(float3) * len));
  float *dev_nhis; // histogram
  checkCudaErrors(hipMalloc((void **) &dev_nhis, sizeof(float) * hist_size));

  // copy host memory to device
  checkCudaErrors(hipMemcpy(dev_data, coords, sizeof(float3) * len,
                             hipMemcpyHostToDevice));

  // initialize histogram with zeros
  hipMemset(dev_nhis, 0.0, hist_size*sizeof(float));

  dim3 numBlocks(8,8,1);
  dim3 threads_per_block(16,16,1);
  gpu_histogram_kernel<<<numBlocks,threads_per_block>>>(dev_data, dev_nhis);

  // copy results from device to host
  checkCudaErrors(hipMemcpy(nhis, dev_nhis, sizeof(float) * hist_size,
                             hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dev_data));
  checkCudaErrors(hipFree(dev_nhis));

}
