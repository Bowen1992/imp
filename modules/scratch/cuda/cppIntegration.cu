#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Example of integrating CUDA functions into an existing
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <IMP/scratch/internal/hip/hip_runtime_api.h>
#include <IMP/scratch/internal/helper_functions.h>

#define SQ(x) (x)*(x)

///////////////////////////////////////////////////////////////////////////////
//! Demonstration that int2 data can be used in the cpp code
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void
kernel(int len, double3 *in, double *out)
{
    // write data to global memor
  const unsigned int i = blockIdx.x;
  //const unsigned int j = threadIdx.y;
  double3 a = in[i];
  for (unsigned int j = 0; j < len; ++j) {
    //const unsigned int j = threadIdx.x;
    double3 b = in[j];
    out[i * len + j] = sqrt(SQ(a.x - b.x) + SQ(a.y - b.y) + SQ(a.z - b.z));
  }
}

extern "C" void
get_distances_gpu(int argc, char ** argv, int len, double3 *coords, double *out)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
  findCudaDevice(1, (const char **)argv);

  // allocate device memory
    double3 *d_data;
    checkCudaErrors(hipMalloc((void **) &d_data, sizeof(double3) * len));
    double *out_data;
    checkCudaErrors(hipMalloc((void **) &out_data, sizeof(double) * len * len));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data, coords, sizeof(double3) * len,
                               hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(out_data, out, sizeof(double) * len * len,
      hipMemcpyHostToDevice));


    dim3 threads(1, 1, 1);
    // setup execution parameters
    dim3 grid(len, 1, 1);

    // execute the kernel
    kernel<<< grid, threads >>>(len, d_data, out_data);

    // copy results from device to host
    checkCudaErrors(hipMemcpy(out, out_data, sizeof(double) * len * len,
                               hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(out_data));

}
